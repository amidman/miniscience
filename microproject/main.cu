
#include <hip/hip_runtime.h>
#include <stdio.h>

const int n = 4096;
const int BlockSize = 16;

__global__ void kernel_down(double *A, double *B, int n, int number_row, int number_column){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by*BlockSize + ty;
    int column = bx*BlockSize + tx; 
    if(number_column < column && number_row < row){
        double glav = A[row*n + number_column]/A[number_row*n + number_column];    
        if(number_column == column){
            B[row] -= B[number_row]*glav;
        }
        A[row*n + column] -= glav*A[number_row*n + column];
    }
}

__global__ void kernel_up(double *A, double *B, int n, int number_row, int number_column){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by*BlockSize + ty;
    int column = bx*BlockSize + tx; 
    if(number_column == column && number_row > row){
        double glav = A[row*n + number_column]/A[number_row*n + number_column];    
        B[row] -= B[number_row]*glav;
        A[row*n + column] = 0;
    }
}

int main (int argc, char* argv []){
    double* A = new double[n * n];
    double* b = new double[n];
    double* result = new double[n];


    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    
    hipEventCreate ( &start );
    hipEventCreate ( &stop );


    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            A[i*n+j] = i+j+1;
        }
        b[i] = i;
    }

    double *dev_a, *dev_b;

    hipMalloc((void**) &dev_a, n*n*sizeof(double));
    hipMalloc((void**) &dev_b, n*sizeof(double));


    hipEventRecord ( start, 0 );
    hipMemcpy(dev_a, A, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n*sizeof(double), hipMemcpyHostToDevice);

    dim3 Grid(n/BlockSize, n/BlockSize);
    dim3 Block(BlockSize, BlockSize);

    for(int i=0;i<n-1;++i){
        kernel_down<<<Grid, Block>>>(dev_a, dev_b, n, i, i);
    }
    for(int i=n-1;i>=0;--i){ 
        kernel_up<<<Grid,Block>>>(dev_a, dev_b, n, i, i);
    }

    hipMemcpy(A, dev_a, n*n*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(b, dev_b, n*sizeof(double), hipMemcpyDeviceToHost);

    for(int i = 0; i < n - 1; ++i){
        result[i] = b[i]/A[i*n + i];
    }


    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );


    printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime );


    hipFree(dev_a);
    hipFree(dev_b);
    hipDeviceReset();

    return 0;
}
